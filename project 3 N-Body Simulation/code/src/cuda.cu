#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>

#ifdef GUI
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#endif

#include "./headers/physics.h"
#include "./headers/checkpoint.h"


int block_size = 1024;


int n_body;
int n_iteration;


__global__ void update_position(double *x, double *y, double *vx, double *vy, int n) {
    //TODO: update position 
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    double dt = 0.0001f;
    
    if (i < n) {
        x[i]+=vx[i]*dt;
        y[i]+=vy[i]*dt;
    }
}

__global__ void update_velocity(double *m, double *x, double *y, double *vx, double *vy, int n) {
    //TODO: calculate force and acceleration, update velocity
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // if (i < n) {  
    // }
    int bound_x = 4000;
    int bound_y = 4000;
    double error = 1e-9f;
    double dt = 0.0001f;
    double gravity_const = 100000.0f;
    double radius2 = 2.0f;

    double tmpax=0,tmpay=0;
    if (i<n){
        for (int j=0;j<n;j++){
        if (i==j)
            continue;
        double dist=pow((x[i]-x[j]),2)+pow((y[i]-y[j]),2);
        if (dist<=radius2){//collision
            continue;
            vx[i]=(vx[i]*(m[i]-m[j])+2*m[j]*vx[j])/(m[i]+m[j]);
            vy[i]=(vy[i]*(m[i]-m[j])+2*m[j]*vy[j])/(m[i]+m[j]);
            vx[j]=(vx[j]*(m[j]-m[i])+2*m[i]*vx[i])/(m[i]+m[j]);
            vy[j]=(vy[j]*(m[j]-m[i])+2*m[i]*vy[i])/(m[i]+m[j]);
            //continue;
        }
        dist+=error;
        double ax=((gravity_const*m[j])/(dist))*((x[j]-x[i])/pow(dist,0.5));
        double ay=((gravity_const*m[j])/(dist))*((y[j]-y[i])/pow(dist,0.5));
        tmpax+=ax;
        tmpay+=ay;
        if (isnan(vx[i]))
            printf("(%d,%lf)",j,dist);
        }
        if (x[i]<=0.1)
            vx[i]=-vx[i];
        if (y[i]<=0.1)
            vy[i]=-vy[i];
        if (x[i]>=0.99*bound_x)
            vx[i]=-vx[i];
        if (y[i]>=0.99*bound_y)
            vy[i]=-vy[i];
        vx[i]+=tmpax*dt;
        vy[i]+=tmpay*dt;
    }

}


void generate_data(double *m, double *x,double *y,double *vx,double *vy, int n) {
    // TODO: Generate proper initial position and mass for better visualization
    srand((unsigned)time(NULL));
    for (int i = 0; i < n; i++) {
        m[i] = rand() % max_mass + 1.0f;
        x[i] = 2000.0f + rand() % (bound_x / 4);
        y[i] = 2000.0f + rand() % (bound_y / 4);
        vx[i] = 0.0f;
        vy[i] = 0.0f;
    }
}


double master() {
    double* m = new double[n_body];
    double* x = new double[n_body];
    double* y = new double[n_body];
    double* vx = new double[n_body];
    double* vy = new double[n_body];
    double totaltime=0;
    generate_data(m, x, y, vx, vy, n_body);

    Logger l = Logger("cuda", n_body, bound_x, bound_y);

    double *device_m;
    double *device_x;
    double *device_y;
    double *device_vx;
    double *device_vy;

    hipMalloc(&device_m, n_body*sizeof(double));
    hipMalloc(&device_x, n_body*sizeof(double));
    hipMalloc(&device_y, n_body*sizeof(double));
    hipMalloc(&device_vx, n_body*sizeof(double));
    hipMalloc(&device_vy, n_body*sizeof(double));

    hipMemcpy(device_m, m, n_body*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_x, x, n_body*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_y, y, n_body*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vx, vx, n_body*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vy, vy, n_body*sizeof(double), hipMemcpyHostToDevice);

    int n_block = n_body / block_size + 1;

    for (int i = 0; i < n_iteration; i++){
        std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

        update_velocity<<<n_block, block_size>>>(device_m, device_x, device_y, device_vx, device_vy, n_body);
        update_position<<<n_block, block_size>>>(device_x, device_y, device_vx, device_vy, n_body);

        hipMemcpy(x, device_x, n_body*sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(y, device_y, n_body*sizeof(double), hipMemcpyDeviceToHost);

        l.save_frame(x, y);

        std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> time_span = t2 - t1;
        totaltime+=time_span.count();
        printf("Iteration %d, elapsed time: %.3f\n", i, time_span.count());

        #ifdef GUI
        glClear(GL_COLOR_BUFFER_BIT);
        glColor3f(1.0f, 0.0f, 0.0f);
        glPointSize(2.0f);
        glBegin(GL_POINTS);
        double xi;
        double yi;
        for (int i = 0; i < n_body; i++){
            xi = x[i];
            yi = y[i];
            glVertex2f(xi, yi);
        }
        glEnd();
        glFlush();
        glutSwapBuffers();
        #else

        #endif

    }

    hipFree(device_m);
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_vx);
    hipFree(device_vy);

    hipFree(device_m);
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_vx);
    hipFree(device_vy);

    delete m;
    delete x;
    delete y;
    delete vx;
    delete vy;
    return totaltime;
}


int main(int argc, char *argv[]){
    
    n_body = atoi(argv[1]);
    n_iteration = atoi(argv[2]);

    #ifdef GUI
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(500, 500);
    glutCreateWindow("N Body Simulation CUDA Implementation");
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    gluOrtho2D(0, bound_x, 0, bound_y);
    #endif

    double time=master();

    printf("Student ID: 119010001\n"); // replace it with your student id
    printf("Name: Your Name\n"); // replace it with your name
    printf("Assignment 2: N Body Simulation CUDA Implementation\n");
    printf("totaltime=%lf",time);
    return 0;

}


