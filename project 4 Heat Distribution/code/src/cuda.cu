#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>

#ifdef GUI
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#endif

#include "./headers/physics.h"
/*
__device__ __managed__ float Threshold=0.00001f;
__device__ __managed__ float Fire_temp=90.0f;
__device__ __managed__ float Wall_temp=0.0f;
__device__ __managed__ float Fire_size=100;
__device__ __managed__ int Resolution=800;
*/

int block_size = 512; // cuda thread block size
int size; // problem size
int iteration=1000;

__global__ void initialize(float *data,int size) {
    // TODO: intialize the temperature distribution (in parallelized way)
    float Wall_temp=0.0f;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= size*size) {
        return;
    }
    data[i] = Wall_temp;
}


__global__ void generate_fire_area(bool *fire_area,int size){
    float Fire_size=100;
    // TODO: generate the fire area (in parallelized way)
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= size*size) {
        return; 
    }
    fire_area[idx]=0;
    float fire1_r2 = Fire_size * Fire_size;
    float fire2_r2 = (Fire_size / 2) * (Fire_size / 2);
    int i=idx/size;
    int j=idx%size;
    int a = i - size / 2;
    int b = j - size / 2;
    int r2 = 0.5 * a * a + 0.8 * b * b - 0.5 * a * b;
    if (r2 < fire1_r2) fire_area[idx] = 1;
    
    int c = i - 1 * size / 3;
    int d = j - 1 * size / 3;
    int r3 = c * c + d * d;
    if (r3 < fire2_r2) fire_area[i * size + j] = 1;
}


__global__ void update(float *data, float *new_data,int size) {
    // TODO: update temperature for each point  (in parallelized way)
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >=size*size) {
        return;
    }
    float total=0;
        int count=0;
        if (i-size>=0){
            total+=data[i-size];
            count++;
        }
        if (i+size<size*size){
            total+=data[i+size];
            count++;
        }
        if (i>=1){
            total+=data[i-1];
            count++;
        }
        if (i+1<size*size){
            total+=data[i+1];
            count++;
        }
        new_data[i]=total/count;
        
    
}


__global__ void maintain_wall(float *data,int size) {
    float Wall_temp=0.0f;
    // TODO: maintain the temperature of the wall (sequential is enough)
    for (int i=0;i<size-1;i++){
        data[i]=Wall_temp;
        data[(i+1)*(size)]=Wall_temp;
        data[(i+1)*(size)+1]=Wall_temp;
        data[(size)*(size-1)+i]=Wall_temp;
    }
}


__global__ void maintain_fire(float *data, bool *fire_area,int size) {
    float Fire_temp=90.0f;
    // TODO: maintain the temperature of the fire (in parallelized way)
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >=size*size) { 
        return; 
    }
    if (fire_area[i])
        data[i]=Fire_temp;
}


#ifdef GUI
__global__ void data2pixels(float *data, GLubyte* pixels,int size){
    int Resolution=800;
    float Fire_temp=90.0f;
    
    // TODO: convert rawdata (large, size^2) to pixels (small, Resolution^2) for faster rendering speed (in parallelized way)
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (idx >= size*size) {
        return; 
    }
    
    int x=idx/Resolution;
    int y=idx%Resolution;
    float factor_data_pixel = (float) size / Resolution;
    float factor_temp_color = (float) 255 / Fire_temp;
    
    
    int idx_pixel = idx * 3;
    int x_raw = x * factor_data_pixel;
    int y_raw = y * factor_data_pixel;
    int idx_raw = y_raw * size + x_raw;
    float temp = data[idx_raw];
    int color =  ((int) temp / 5 * 5) * factor_temp_color;
    pixels[idx_pixel] = color;
    pixels[idx_pixel + 1] = 255 - color;
    pixels[idx_pixel + 2] = 255 - color;
    
}


void plot(GLubyte* pixels){
    // visualize temprature distribution
    #ifdef GUI
    int Resolution=800;
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawPixels(Resolution, Resolution, GL_RGB, GL_UNSIGNED_BYTE, pixels);
    glutSwapBuffers();
    #endif
}
#endif


void master() {
    int Resolution=800;
    float *data_odd;
    float *data_even;
    bool *fire_area;

    hipMalloc(&data_odd, size * size * sizeof(float));
    hipMalloc(&data_even, size * size * sizeof(float));
    hipMalloc(&fire_area, size * size * sizeof(bool));
    //hipMemcpyToSymbol(HIP_SYMBOL(size),&size,sizeof(float));
    #ifdef GUI
    GLubyte *pixels;
    GLubyte *host_pixels;
    host_pixels = new GLubyte[Resolution * Resolution * 3];
    hipMalloc(&pixels, resolution * resolution * 3 * sizeof(GLubyte));
    #endif

    int n_block_size = size * size / block_size + 1;
    int n_block_resolution = Resolution * Resolution / block_size + 1;

    initialize<<<n_block_size, block_size>>>(data_odd,size);
    generate_fire_area<<<n_block_size, block_size>>>(fire_area,size);
    
    int count = 1;
    double total_time = 0;

    while (true){
        std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

        // TODO: modify the following lines to fit your need.
        if (count % 2 == 1) {
            update<<<n_block_size, block_size>>>(data_odd, data_even,size);
            maintain_fire<<<n_block_size, block_size>>>(data_even, fire_area,size);
            maintain_wall<<<1, 1>>>(data_even,size);
        } else {
            update<<<n_block_size, block_size>>>(data_even, data_odd,size);
            maintain_fire<<<n_block_size, block_size>>>(data_odd, fire_area,size);
            maintain_wall<<<1, 1>>>(data_odd,size);
        }

        std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
        double this_time = std::chrono::duration<double>(t2 - t1).count();
        total_time += this_time;
        //printf("Iteration %d, elapsed time: %.6f\n", count, this_time);
        count++;
        
        #ifdef GUI
        if (count % 2 == 1) {
            data2pixels<<<n_block_resolution, block_size>>>(data_even, pixels,size);
        } else {
            data2pixels<<<n_block_resolution, block_size>>>(data_odd, pixels,size);
        }
        if (count==iteration)
            break;
        hipMemcpy(host_pixels, pixels, resolution * resolution * 3 * sizeof(GLubyte), hipMemcpyDeviceToHost);
        //printf("%d",host_pixels[100]);
        plot(host_pixels);
        #endif

    }

    //printf("Converge after %d iterations, elapsed time: %.6f, average computation time: %.6f\n", count-1, total_time, (double) total_time / (count-1));
    printf("%d %d %.6f %.6f\n",size, count-1, total_time, (double) total_time / (count-1));

    hipFree(data_odd);
    hipFree(data_even);
    hipFree(fire_area);

    #ifdef GUI
    hipFree(pixels);
    delete[] host_pixels;
    #endif
    
}


int main(int argc, char *argv[]){
    
    size = atoi(argv[1]);
    
    #ifdef GUI
    int Resolution=800;
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(Resolution, Resolution);
    glutCreateWindow("Heat Distribution Simulation Sequential Implementation");
    gluOrtho2D(0, Resolution, 0, Resolution);
    #endif

    master();

    //printf("Student ID: 121090327\n"); // replace it with your student id
    //printf("Name: Zijuan Lin\n"); // replace it with your name
    //printf("Assignment 4: Heat Distribution CUDA Implementation\n");

    return 0;

}


